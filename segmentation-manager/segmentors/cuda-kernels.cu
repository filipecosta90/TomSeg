#include <hip/hip_runtime.h>

#include <stdio.h>

__global__
void zero_GPU( double *l_p_array_gpu ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // <-- in case you use more blocks
    printf("  %i: Hello World!\n", i);
    l_p_array_gpu[i] = 0;
}

void zero(double *l_p_array, int a_numElements)
{
    double *l_p_array_gpu;

    int size = a_numElements * int(sizeof(double));

    hipMalloc((void**) &l_p_array_gpu, size);

    hipMemcpy(l_p_array_gpu, l_p_array, size, hipMemcpyHostToDevice);

    // use one block with a_numElements threads
    zero_GPU<<<1, a_numElements>>>(l_p_array_gpu);

    hipMemcpy(l_p_array, l_p_array_gpu, size, hipMemcpyDeviceToHost);

    hipFree(l_p_array_gpu);
}
