#include <hip/hip_runtime.h>


#define CUDA_DEBUG 0

#if CUDA_DEBUG
#include <stdio.h>
#endif

__global__
void erode_GPU( const unsigned char *labels, unsigned char *res, int size, int width, int height ) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if( id > width * height ) return;

    int i = id / width;
    int j = id % width;

    int color = labels[id];

    size = size / 2;

    bool fits = true;

    for( int y = max(0, i - size); fits && y < min(height, i + size); y++ ) {
        for( int x = max(0, j - size); fits && x < min(width, j + size); x++ ) {
            fits = labels[y * width + x] == color;
        }
    }

    if( !fits ) {
        res[id] = 255;
    } else {
        res[id] = color;
    }
}

__global__
void dilate_GPU( const unsigned char *labels, unsigned char *res, int size, int width, int height ) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if( id > width * height ) return;

    int i = id / width;
    int j = id % width;

    int color = 255;

    size = size / 2;

    for( int y = max(0, i - size); y < min(height, i + size); y++ ) {
        for( int x = max(0, j - size); x < min(width, j + size); x++ ) {
            color = min(color, labels[y * width + x]);
        }
    }

    res[id] = color;
}

void erodeAndDilate_GPU( unsigned char* labels, int size, int width, int height ) {
    unsigned char  *labels_gpu, *res;
    int sizeLabels_bytes = width * height * int(sizeof(unsigned char));

#if CUDA_DEBUG
    float millisecondsSending = 0;
    float millisecondsKernel = 0;
    float millisecondsReceiving = 0;
    float milliseconds = 0;
    cudaEvent_t start, stop, startSending, stopSending, startKernel, stopKernel, startReceiving, stopReceiving;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);
    cudaEventCreate(&startSending);
    cudaEventCreate(&stopSending);
    cudaEventCreate(&startKernel);
    cudaEventCreate(&stopKernel);
    cudaEventCreate(&startReceiving);
    cudaEventCreate(&stopReceiving);

    cudaEventRecord(start);
#endif
    hipMalloc((void**) &labels_gpu, sizeLabels_bytes);
    hipMalloc((void**) &res, sizeLabels_bytes);
#if CUDA_DEBUG
    cudaEventRecord(startSending);
#endif
    hipMemcpy(labels_gpu, labels, sizeLabels_bytes, hipMemcpyHostToDevice);
#if CUDA_DEBUG
    cudaEventRecord(stopSending);
    cudaEventSynchronize(stopSending);
    cudaEventElapsedTime(&millisecondsSending, startSending, stopSending);

    cudaEventRecord(startKernel);
#endif
    size_t numThreads = 512;
    size_t numBloks = (width*height) / numThreads;
    erode_GPU<<<numBloks, numThreads>>>(labels_gpu, res, size, width, height);
    dilate_GPU<<<numBloks, numThreads>>>(res, labels_gpu, size, width, height);
#if CUDA_DEBUG
    cudaEventRecord(stopKernel);
    cudaEventSynchronize(stopKernel);
    cudaEventElapsedTime(&millisecondsKernel, startKernel, stopKernel);

    cudaEventRecord(startReceiving);
#endif
    hipMemcpy(labels, labels_gpu, sizeLabels_bytes, hipMemcpyDeviceToHost);
#if CUDA_DEBUG
    cudaEventRecord(stopReceiving);
    cudaEventSynchronize(stopReceiving);
    cudaEventElapsedTime(&millisecondsReceiving, startReceiving, stopReceiving);
#endif
    hipFree(labels_gpu);
    hipFree(res);
#if CUDA_DEBUG
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&milliseconds, start, stop);

    printf("Sending Time:\t%f ms\n", millisecondsSending);
    printf("Kernel Time:\t%f ms\n", millisecondsKernel);
    printf("Receiving Time:\t%f ms\n", millisecondsReceiving);
    printf("Overall Time:\t%f ms\n", milliseconds);
#endif
}
