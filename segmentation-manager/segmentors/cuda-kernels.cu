#include <hip/hip_runtime.h>


#include <stdio.h>

__global__
void erode_GPU( const unsigned char *labels, unsigned char *res, int size, int width, int height ) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if( id > width * height ) return;

    int i = id / width;
    int j = id % width;

    int color = labels[id];

    size = size / 2;

    bool fits = true;

    for( int y = max(0, i - size); fits && y < min(height, i + size); y++ ) {
        for( int x = max(0, j - size); fits && x < min(width, j + size); x++ ) {
            fits = labels[y * width + x] == color;
        }
    }

    if( !fits ) {
        res[id] = 255;
    } else {
        res[id] = color;
    }
}

__global__
void dilate_GPU( const unsigned char *labels, unsigned char *res, int size, int width, int height ) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if( id > width * height ) return;

    int i = id / width;
    int j = id % width;

    int color = 255;

    size = size / 2;

    for( int y = max(0, i - size); y < min(height, i + size); y++ ) {
        for( int x = max(0, j - size); x < min(width, j + size); x++ ) {
            color = min(color, labels[y * width + x]);
        }
    }

    res[id] = color;
}

void erodeAndDilate_GPU( unsigned char* labels, int size, int width, int height ) {
    unsigned char  *labels_gpu, *res;
    int sizeLabels_bytes = width * height * int(sizeof(unsigned char));

    hipMalloc((void**) &labels_gpu, sizeLabels_bytes);
    hipMalloc((void**) &res, sizeLabels_bytes);

    hipMemcpy(labels_gpu, labels, sizeLabels_bytes, hipMemcpyHostToDevice);

    size_t numThreads = 512;
    size_t numBloks = (width*height) / numThreads;
    erode_GPU<<<numBloks, numThreads>>>(labels_gpu, res, size, width, height);
    dilate_GPU<<<numBloks, numThreads>>>(res, labels_gpu, size, width, height);

    hipMemcpy(labels, labels_gpu, sizeLabels_bytes, hipMemcpyDeviceToHost);

    hipFree(labels_gpu);
    hipFree(res);
}
